#include "hip/hip_runtime.h"
/**
 * @file CudaMath.cu
 * @author Bryn McKerracher
 * @date 20/10/2021
 * @brief Contains Cuda array math for matrix operations.
 */

#include "CudaMath.h"

/**
 * @brief Returns an epsilon value for either float or double.
 * @tparam T The floating point type to get epsilon for.
 * @return The epsilon value for the given floating point type.
 */
template <typename T>
static __device__ T epsilon();

template<> __device__ float epsilon<float>() {
    return FLT_EPSILON;
}

template<> __device__ double epsilon<double>() {
    return DBL_EPSILON;
}

const unsigned BlockSize = 1024; ///< Number of threads per GPU block.

/**
 * @brief Gets the num of thread blocks for a given CUDA array size.
 * @param n The size of the array to get thread blocks sizing for.
 * @return The number of thread blocks for a CUDA operation on an array of size n.
 */
inline size_t GetNumBlocks(size_t n) {
    return (n + BlockSize - 1) / BlockSize;
}

/**
 * Function expects that all arrays have already been allocated and that each array has n elements.
 * No bounds checking is performed.
 *
 * @brief Performs an add operation on each pair of elements from a and b, stores the result in sum.
 * @tparam T The type of elements in the arrays.
 * @param n The number of elements in each array.
 * @param a Pointer to the first summand.
 * @param b Pointer to the second summand.
 * @param sum Pointer to the sum.
 */
template <typename T>
__global__ void deviceAdd(size_t n, T* a, T* b, T* sum) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        sum[i] = a[i] + b[i];
    }
}

/**
 * Function expects that all arrays have already been allocated and that each array has n elements.
 * No bounds checking is performed.
 *
 * @brief Performs a subtraction operation on each pair of elements from a and b, stores the result in result.
 * @tparam T The type of elements in the arrays.
 * @param n The number of elements in each array.
 * @param a Pointer to the minuend.
 * @param b Pointer to the second subtrahend.
 * @param result Pointer to the results array.
 */
template <typename T>
__global__ void deviceSubtract(size_t n, T* a, T* b, T* result) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        result[i] = a[i] - b[i];
    }
}

/**
 * Function expects that all arrays have already been allocated and that each array has n elements.
 * No bounds checking is performed.
 *
 * @brief Performs pointwise scalar multiplication of a by b, and stores the results in result.
 * @tparam T The type of the elements in a, either float or double.
 * @param n The size of the array in a.
 * @param a Pointer to the array being multiplied.
 * @param b Scalar to multiply by.
 * @param result Pointer to result array.
 */
template <typename T>
__global__ void deviceScalarMultiply(size_t n, T* a, T b, T* result) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        result[i] = a[i] * b;
    }
}

/**
 * @brief Performs an element-wise comparison between the provided arrays, and sets equal accordingly.
 * @tparam T The types of elements of each array.
 * @param n The number of elements in each array.
 * @param a The first array.
 * @param b The second array.
 * @param equal Pointer to the result flag.
 */
template <typename T>
__global__ void deviceEqual(size_t n, T* a, T* b, bool* equal) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if ((a[i] > b[i] ? a[i] - b[i] : b[i] - a[i]) > (a[i] < b[i] ? a[i] : b[i]) * epsilon<T>()) {
            *equal = false;
            return;
        }
    }
}

/**
 * @brief Calculates the transpose of a matrix.
 * @tparam T The types of elements in the matrix.
 * @param n The number of elements in the matrix.
 * @param oldWidth The original width of the matrix.
 * @param a Pointer to the entries of the matrix.
 * @param result Pointer to the results array of the matrix.
 */
template <typename T>
__global__ void deviceTranspose(size_t n, size_t oldWidth, T* a, T* result) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        const size_t x = i % oldWidth;
        const size_t y = i / oldWidth;
        result[y + x * (n / oldWidth)] = a[i];
    }
}

template <typename T>
__global__ void deviceZeroArray(size_t n, T* a) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        a[i] = 0;
    }
}

template <typename T>
__global__ void deviceIdentityArray(size_t n, size_t width, T* a) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        a[i] = (i % width == i / width) ? 1 : 0;
    }
}

template <typename T>
void CudaMath::cudaAdd(size_t n, T *a, T *b, T *sum) {
    deviceAdd<T><<<GetNumBlocks(n), BlockSize>>>(n, a, b, sum);
    hipDeviceSynchronize();
}

template <typename T>
void CudaMath::cudaSubtract(size_t n, T *a, T *b, T *result) {
    deviceSubtract<T><<<GetNumBlocks(n), BlockSize>>>(n, a, b, result);
    hipDeviceSynchronize();
}

template <typename T>
void CudaMath::cudaScalarMultiply(size_t n, T *a, T b, T *result) {
    deviceScalarMultiply<<<GetNumBlocks(n), BlockSize>>>(n, a, b, result);
    hipDeviceSynchronize();
}

template <typename T>
bool CudaMath::cudaEqual(size_t n, T *a, T *b) {
    //Allocate bool
    bool* deviceEqualFlag = nullptr;
    hipMallocManaged(&deviceEqualFlag, sizeof (bool));
    *deviceEqualFlag = true;
    //Create host flag
    bool hostEqualFlag = false;
    //Run CUDA check and sync threads
    deviceEqual<T><<<GetNumBlocks(n), BlockSize>>>(n, a, b, deviceEqualFlag);
    hipDeviceSynchronize();
    //Copy flag from device to host
    hipMemcpy(&hostEqualFlag, deviceEqualFlag, sizeof (bool), hipMemcpyDeviceToHost);
    //Free memory
    hipFree(deviceEqualFlag);
    return hostEqualFlag;
}

template <typename T>
void CudaMath::cudaTranspose(size_t n, size_t oldWidth, T* a, T* result) {
    deviceTranspose<T><<<GetNumBlocks(n), BlockSize>>>(n, oldWidth, a, result);
    hipDeviceSynchronize();
}

template <typename T>
void CudaMath::cudaZeroArray(size_t n, T *a) {
    deviceZeroArray<<<GetNumBlocks(n), BlockSize>>>(n, a);
    hipDeviceSynchronize();
}

template <typename T>
void CudaMath::cudaIdentityArray(size_t n, size_t width, T *a) {
    deviceIdentityArray<<<GetNumBlocks(n), BlockSize>>>(n, width, a);
    hipDeviceSynchronize();
}

///Forward declarations
//Add
template void CudaMath::cudaAdd<float>(size_t n, float* a, float* b, float* sum);
template void CudaMath::cudaAdd<double>(size_t n, double* a, double* b, double* sum);

//Subtract
template void CudaMath::cudaSubtract<float>(size_t n, float* a, float* b, float* result);
template void CudaMath::cudaSubtract<double>(size_t n, double* a, double* b, double* result);

//Scalar multiply
template void CudaMath::cudaScalarMultiply<float>(size_t n, float* a, float b, float* result);
template void CudaMath::cudaScalarMultiply<double>(size_t n, double* a, double b, double* result);

//Equal
template bool CudaMath::cudaEqual<float>(size_t n, float* a, float* b);
template bool CudaMath::cudaEqual<double>(size_t n, double* a, double* b);

//Transpose
template void CudaMath::cudaTranspose<float>(size_t n, size_t oldWidth, float* a, float* result);
template void CudaMath::cudaTranspose<double>(size_t n, size_t oldWidth, double* a, double* result);

//Zero array
template void CudaMath::cudaZeroArray<float>(size_t n, float* a);
template void CudaMath::cudaZeroArray<double>(size_t n, double* a);

//Identity array
template void CudaMath::cudaIdentityArray<float>(size_t n, size_t width, float* a);
template void CudaMath::cudaIdentityArray<double>(size_t n, size_t width, double* a);